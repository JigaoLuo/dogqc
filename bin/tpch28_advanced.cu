#include "hip/hip_runtime.h"
/// select l_suppkey, count(*)
/// from lineitem
/// group by l_suppkey
/// order by l_suppkey
#include <map>
#include <cassert>
//#define COLLISION_PRINT
//#define HT_CHECKER

#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct apayl2 {
    int att4_lsuppkey;
};

__device__ bool operator==(const apayl2& lhs, const apayl2& rhs) {
    return lhs.att4_lsuppkey == rhs.att4_lsuppkey;
}

constexpr int SHARED_MEMORY_HT_SIZE = 1024;  /// In shared memory
constexpr int LINEITEM_SIZE = 6001215;       /// SF1
//constexpr int LINEITEM_SIZE = 59986052;      /// SF10, change the folder name to sf10
constexpr int GLOBAL_HT_SIZE = LINEITEM_SIZE * 2;  /// In global memory
//constexpr int GLOBAL_HT_SIZE = 65536;  /// In global memory

__device__ void sm_to_gm(agg_ht_sm<apayl2>* aht2, int* agg1, agg_ht<apayl2>* g_aht2, int* g_agg1) {
    /// Copy the shared memory hash table (pre-aggreagation) into the global hash table.
    {
        /// <-- START: first half of the kernel 2
        int att4_lsuppkey;
        int att1_countlsu;
        int tid_aggregation2 = 0;
        unsigned loopVar = threadIdx.x;  ///
        unsigned step = blockDim.x;  ///
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_aggregation2 = loopVar;
            active = (loopVar < SHARED_MEMORY_HT_SIZE);  ///
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
            }
            // -------- scan aggregation ht (opId: 2) --------
            if(active) {
                active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
            }
            if(active) {
                apayl2 payl = aht2[tid_aggregation2].payload;
                att4_lsuppkey = payl.att4_lsuppkey;
            }
            if(active) {
                att1_countlsu = agg1[tid_aggregation2];
            }
            /// <-- END: first half of the kernel 2

            /// <-- START: second half of the kernel 1
            /// Insert to global hash table.
            int bucket = 0;
            if(active) {
                uint64_t hash2 = 0;
                hash2 = 0;
                if(active) {
                    hash2 = hash ( (hash2 + ((uint64_t)att4_lsuppkey)));
                }
                apayl2 payl;
                payl.att4_lsuppkey = att4_lsuppkey;
                int bucketFound = 0;
                int numLookups = 0;
                while(!(bucketFound)) {
                    bucket = hashAggregateGetBucket ( g_aht2, GLOBAL_HT_SIZE, hash2, numLookups, &(payl));  ////
                    apayl2 probepayl = g_aht2[bucket].payload;  ////
                    bucketFound = 1;
                    bucketFound &= ((payl.att4_lsuppkey == probepayl.att4_lsuppkey));
                }
            }
            if(active) {
                atomicAdd(&(g_agg1[bucket]), ((int)att1_countlsu));  ////
            }
            /// <-- END: second half of the kernel 1
            loopVar += step;
        }
    }
}

__global__ void krnl_lineitem1(
    int* iatt4_lsuppkey, int* nout_result, int* oatt4_lsuppkey, int* oatt1_countlsu, agg_ht<apayl2>* g_aht2, int* g_agg1) {  ///

    /// local block memory cache : ONLY FOR A BLOCK'S THREADS!!!
    extern __shared__ char shared_memory[];
    agg_ht_sm<apayl2>* aht2 = (agg_ht_sm<apayl2> *)shared_memory;  ///
    int* agg1 = (int*)(shared_memory + sizeof(agg_ht_sm<apayl2>) * SHARED_MEMORY_HT_SIZE);  ///
    volatile __shared__ int HT_FULL_FLAG; HT_FULL_FLAG = 0;  ////
#ifdef COLLISION_PRINT
    __shared__ int num_collision; num_collision = 0;
#endif

    initSMAggHT(aht2,SHARED_MEMORY_HT_SIZE);
    initSMAggArray(agg1,SHARED_MEMORY_HT_SIZE);
    __syncthreads();

    {
        /// The first old kenrel
        int att4_lsuppkey;

        int tid_lineitem1 = 0;
        unsigned loopVar__ = ((blockIdx.x * blockDim.x) + threadIdx.x);  ////
//        unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned step = (blockDim.x * gridDim.x);
        unsigned flushPipeline__ = 0;  ////
//        unsigned flushPipeline = 0;
        int active = 0;
//        while(!(flushPipeline)) {
        while(!(flushPipeline__)) {
            tid_lineitem1 = loopVar__;
            active = (loopVar__ < LINEITEM_SIZE);
            // flush pipeline if no new elements
            flushPipeline__ = !(__ballot_sync(ALL_LANES,active));
            if(active) {
                att4_lsuppkey = iatt4_lsuppkey[tid_lineitem1];
            }
            // -------- aggregation (opId: 2) --------
            int bucket = 0;
            if(active) {
                uint64_t hash2 = 0;
                hash2 = 0;
                if(active) {
                    hash2 = hash ( (hash2 + ((uint64_t)att4_lsuppkey)));
                }
                apayl2 payl;
                payl.att4_lsuppkey = att4_lsuppkey;
                int bucketFound = 0;
                int numLookups = 0;
                while(!(bucketFound)) {   ////
                    bucket = hashAggregateGetBucket ( aht2, SHARED_MEMORY_HT_SIZE, hash2, numLookups, &(payl));  ///
                    if (bucket != -1) {  ////
                        apayl2 probepayl = aht2[bucket].payload;
                        bucketFound = 1;
                        bucketFound &= ((payl.att4_lsuppkey == probepayl.att4_lsuppkey));
                    } else {  ////
                        assert(bucketFound == 0);  ////
                        loopVar__ -= step;
                        atomicAdd((int *)&HT_FULL_FLAG, 1);  ////
                        break;  ////
                    }  ////
                }
#ifdef COLLISION_PRINT
                atomicAdd(&num_collision, numLookups - 1);
#endif
            }
            if(active && bucket != -1) {  ////
                atomicAdd(&(agg1[bucket]), ((int)1));
            }

            /// Implication and Disjunction: P->Q <=>  ^P OR Q
            /// bucket==-1 -> HT_FULL_FLAG!=0
            assert(bucket != -1 || HT_FULL_FLAG!=0);

            __syncthreads();  ////
            if (HT_FULL_FLAG != 0) {
                sm_to_gm(aht2, agg1, g_aht2, g_agg1);
                __threadfence_block(); /// Ensure the ordering:
                initSMAggHT(aht2,SHARED_MEMORY_HT_SIZE);
                initSMAggArray(agg1,SHARED_MEMORY_HT_SIZE);
                if (threadIdx.x == 0) HT_FULL_FLAG = 0;
                __syncthreads();  ////
            }
            loopVar__ += step;
        }
    }


    __syncthreads();  ///
#ifdef COLLISION_PRINT
    if (threadIdx.x == 0) {
        /// Allow only one print per block here.
        printf("In Block %d: num_collision: %d\n", blockIdx.x, num_collision);
    }
#endif

#ifdef HT_CHECKER
    if (threadIdx.x == 0) {
        if (HT_FULL_FLAG != 0) {
            printf("FUll.\n");
        } else {
            printf("Not FULL.\n");
        }
    }
#endif
    sm_to_gm(aht2, agg1, g_aht2, g_agg1);
}

__global__ void krnl_aggregation2(
    agg_ht<apayl2>* aht2, int* agg1, int* nout_result, int* oatt4_lsuppkey, int* oatt1_countlsu) {
    int att4_lsuppkey;
    int att1_countlsu;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation2 = loopVar;
        active = (loopVar < GLOBAL_HT_SIZE);  ///
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 2) --------
        if(active) {
            active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl2 payl = aht2[tid_aggregation2].payload;
            att4_lsuppkey = payl.att4_lsuppkey;
        }
        if(active) {
            att1_countlsu = agg1[tid_aggregation2];
        }
        // -------- projection (no code) (opId: 3) --------
        // -------- materialize (opId: 4) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt4_lsuppkey[wp] = att4_lsuppkey;
            oatt1_countlsu[wp] = att1_countlsu;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt4_lsuppkey;
    iatt4_lsuppkey = ( int*) map_memory_file ( "mmdb/tpch-dbgen-sf1/lineitem_l_suppkey" );

    int nout_result;
    std::vector < int > oatt4_lsuppkey(LINEITEM_SIZE);
    std::vector < int > oatt1_countlsu(LINEITEM_SIZE);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt4_lsuppkey;
    hipMalloc((void**) &d_iatt4_lsuppkey, LINEITEM_SIZE* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt4_lsuppkey;
    hipMalloc((void**) &d_oatt4_lsuppkey, LINEITEM_SIZE* sizeof(int) );
    int* d_oatt1_countlsu;
    hipMalloc((void**) &d_oatt1_countlsu, LINEITEM_SIZE* sizeof(int) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    agg_ht<apayl2>* d_aht2;
    hipMalloc((void**) &d_aht2, GLOBAL_HT_SIZE* sizeof(agg_ht<apayl2>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht2, GLOBAL_HT_SIZE);
    }
    int* d_agg1;
    hipMalloc((void**) &d_agg1, GLOBAL_HT_SIZE* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0, GLOBAL_HT_SIZE);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt4_lsuppkey, iatt4_lsuppkey, LINEITEM_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    std::clock_t start_krnl_lineitem11 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        const int shared_memory_usage = (sizeof(agg_ht_sm<apayl2>) + sizeof(int)) * SHARED_MEMORY_HT_SIZE;
        std::cout << "Shared memory usage: " << shared_memory_usage << " bytes" << std::endl;
        hipFuncSetAttribute(reinterpret_cast<const void*>(krnl_lineitem1), hipFuncAttributeMaxDynamicSharedMemorySize, /*65536*/ shared_memory_usage);
        krnl_lineitem1<<<gridsize, blocksize, shared_memory_usage>>>(d_iatt4_lsuppkey, d_nout_result, d_oatt4_lsuppkey, d_oatt1_countlsu, d_aht2, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem11 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }

    std::clock_t start_krnl_aggregation22 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation2<<<gridsize, blocksize>>>(d_aht2, d_agg1, d_nout_result, d_oatt4_lsuppkey, d_oatt1_countlsu);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation22 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation2! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation2")
        }
    }
    std::clock_t stop_totalKernelTime0 = std::clock();

    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt4_lsuppkey.data(), d_oatt4_lsuppkey, LINEITEM_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_countlsu.data(), d_oatt1_countlsu, LINEITEM_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt4_lsuppkey);
    hipFree( d_aht2);
    hipFree( d_agg1);
    hipFree( d_nout_result);
    hipFree( d_oatt4_lsuppkey);
    hipFree( d_oatt1_countlsu);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }


    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > LINEITEM_SIZE)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("l_suppkey: ");
        printf("%8i", oatt4_lsuppkey[pv]);
        printf("  ");
        printf("count_l_suppkey: ");
        printf("%8i", oatt1_countlsu[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {  ///
        printf("[...]\n");
    }
    printf("\n");

    /// 40 sorted output
    std::map<int, int> ht;
    for ( int pv = 0; pv < nout_result; pv += 1) {
        ht.emplace(oatt4_lsuppkey[pv], oatt1_countlsu[pv]);
    }
    auto it = ht.begin();
    printf("\nSorted Result: %ld tuples\n", ht.size());
    for ( int pv = 0; ((pv < 25) && (pv < ht.size())); pv += 1) {
        printf("l_orderkey: ");
        printf("%8i", it->first);
        printf("  ");
        printf("count_l_orderkey: ");
        printf("%8i", it->second);
        printf("  ");
        printf("\n");
        std::advance(it, 1);
    }
    if((ht.size() > 25)) {
        printf("[...sorted...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();


    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem1", (stop_krnl_lineitem11 - start_krnl_lineitem11) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation2", (stop_krnl_aggregation22 - start_krnl_aggregation22) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
