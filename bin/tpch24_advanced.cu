#include "hip/hip_runtime.h"
/// My Query 24
/// select l_linenumber --> l_linenumber is the 4th attribute in lineitem table
/// from lineitem
/// group by l_linenumber
#include <unordered_set>
#include <cassert>
#define COLLISION_PRINT

#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct apayl2 {
    int att4_llinenum;
};

constexpr int SHARED_MEMORY_SIZE = 49152;  /// Total amount of shared memory per block:       49152 bytes

__global__ void krnl_lineitem1(
    int* iatt4_llinenum, int* nout_result, int* oatt4_llinenum) {

    /// local block memory cache : ONLY FOR A BLOCK'S THREADS!!!
    const int HT_SIZE = 128;
    __shared__ agg_ht<apayl2> aht2[HT_SIZE];  ///
#ifdef COLLISION_PRINT
    __shared__ int num_collision; num_collision = 0;
#endif
    const int shared_memory_usage = sizeof(aht2);
    assert(shared_memory_usage <= SHARED_MEMORY_SIZE);  /// Check stuff fits into shared memory in a SM.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        /// Allow only one print here.
        printf("Shared memory usage: %d / %d bytes.\n", shared_memory_usage, SHARED_MEMORY_SIZE);
    }

    {
        /// Init hash table in shared memory.
        int ht_index;
        unsigned loopVar = threadIdx.x;  ///
        unsigned step = blockDim.x;  ///
        while(loopVar < HT_SIZE) {
            ht_index = loopVar;
            aht2[ht_index].lock.init();
            aht2[ht_index].hash = HASH_EMPTY;
            loopVar += step;
        }
    }

    __syncthreads();

    {
        /// The first old kenrel
        int att4_llinenum;

        int tid_lineitem1 = 0;
        unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned step = (blockDim.x * gridDim.x);
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_lineitem1 = loopVar;
            active = (loopVar < 6001215);
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
                att4_llinenum = iatt4_llinenum[tid_lineitem1];
            }
            // -------- aggregation (opId: 2) --------
            int bucket = 0;
            if(active) {
                uint64_t hash2 = 0;
                hash2 = 0;
                if(active) {
                    hash2 = hash ( (hash2 + ((uint64_t)att4_llinenum)));
                }
                apayl2 payl;
                payl.att4_llinenum = att4_llinenum;
                int bucketFound = 0;
                int numLookups = 0;
                while(!(bucketFound)) {
                    bucket = hashAggregateGetBucket ( aht2, HT_SIZE, hash2, numLookups, &(payl));  ///
                    apayl2 probepayl = aht2[bucket].payload;
                    bucketFound = 1;
                    bucketFound &= ((payl.att4_llinenum == probepayl.att4_llinenum));
                }
#ifdef COLLISION_PRINT
                atomicAdd(&num_collision, numLookups - 1);
#endif
            }
            if(active) {
            }
            loopVar += step;
        }
    }

    __syncthreads();  ///
#ifdef COLLISION_PRINT
    if (threadIdx.x == 0) {
        /// Allow only one print per block here.
        printf("In Block %d: num_collision: %d\n", blockIdx.x, num_collision);
    }
#endif

    {
        /// The second old kernel
        int att4_llinenum;
        unsigned warplane = (threadIdx.x % 32);
        unsigned prefixlanes = (0xffffffff >> (32 - warplane));
        int tid_aggregation2 = 0;
//        unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned loopVar = threadIdx.x;  ///
//        unsigned step = (blockDim.x * gridDim.x);
        unsigned step = blockDim.x;  ///
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_aggregation2 = loopVar;
            active = (loopVar < HT_SIZE);  ///
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
            }
            // -------- scan aggregation ht (opId: 2) --------
            if(active) {
                active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
            }
            if(active) {
                apayl2 payl = aht2[tid_aggregation2].payload;
                att4_llinenum = payl.att4_llinenum;
            }
            if(active) {
            }
            // -------- projection (no code) (opId: 3) --------
            // -------- materialize (opId: 4) --------
            int wp;
            int writeMask;
            int numProj;
            writeMask = __ballot_sync(ALL_LANES,active);
            numProj = __popc(writeMask);
            if((warplane == 0)) {
                wp = atomicAdd(nout_result, numProj);
            }
            wp = __shfl_sync(ALL_LANES,wp,0);
            wp = (wp + __popc((writeMask & prefixlanes)));
            if(active) {
                oatt4_llinenum[wp] = att4_llinenum;
            }
            loopVar += step;
        }
    }
}

__global__ void krnl_reduce1(  ///
        int* oatt4_llinenum, agg_ht<apayl2>* aht2, int* nout_result) {  ///
    int att4_llinenum;  ///

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < *nout_result);  ///
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att4_llinenum = oatt4_llinenum[tid_lineitem1];  ///
        }
        // -------- aggregation (opId: 2) --------
        int bucket = 0;
        if(active) {
            uint64_t hash2 = 0;
            hash2 = 0;
            if(active) {
                hash2 = hash ( (hash2 + ((uint64_t)att4_llinenum)));
            }
            apayl2 payl;
            payl.att4_llinenum = att4_llinenum;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( aht2, *nout_result * 2, hash2, numLookups, &(payl));  ///
                apayl2 probepayl = aht2[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl.att4_llinenum == probepayl.att4_llinenum));
            }
        }
        if(active) {
        }
        loopVar += step;
    }
}

__global__ void krnl_reduce2(
        agg_ht<apayl2>* aht2, int* n_final_out_result, int* oatt4_llinenum, int* nout_result) {
    int att4_llinenum;  ///

    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));
    int tid_aggregation2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation2 = loopVar;
        active = (loopVar < *nout_result * 2);  ///
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 2) --------
        if(active) {
            active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl2 payl = aht2[tid_aggregation2].payload;
            att4_llinenum = payl.att4_llinenum;
        }
        if(active) {
        }
        // -------- projection (no code) (opId: 3) --------
        // -------- materialize (opId: 4) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(n_final_out_result, numProj);  ///
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt4_llinenum[wp] = att4_llinenum;
        }
        loopVar += step;
    }
}

int main() {
    int* iatt4_llinenum;
    iatt4_llinenum = ( int*) map_memory_file ( "mmdb/lineitem_l_linenumber" );

    int nout_result;
    int final_nout_result;  ///
    std::vector < int > oatt4_llinenum(6001215);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt4_llinenum;
    hipMalloc((void**) &d_iatt4_llinenum, 6001215* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_final_nout_result;  ///
    hipMalloc((void**) &d_final_nout_result, 1* sizeof(int) );  ///
    int* d_oatt4_llinenum;
    hipMalloc((void**) &d_oatt4_llinenum, 6001215* sizeof(int) );
    int* d_final_oatt4_llinenum;  ///
    hipMalloc((void**) &d_final_oatt4_llinenum, 6001215* sizeof(int) );  ///
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt4_llinenum, iatt4_llinenum, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem1<<<gridsize, blocksize>>>(d_iatt4_llinenum, d_nout_result, d_oatt4_llinenum);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }
    std::clock_t stop_totalKernelTime0 = std::clock();





    ///
    // input: d_oatt4_llinenum
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy nout_result! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }


    /// output size is less than d_nout_result OR nout_result
    agg_ht<apayl2>* d_aht2;
    hipMalloc((void**) &d_aht2, nout_result * 2 * sizeof(agg_ht<apayl2>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht2, nout_result * 2 );
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_final_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }

    std::clock_t start_krnl_reduce1 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_reduce1<<<gridsize, blocksize>>>(d_oatt4_llinenum, d_aht2, d_nout_result);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_reduce1 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_reduce1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }

    std::clock_t start_krnl_reduce2 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_reduce2<<<gridsize, blocksize>>>(d_aht2, d_final_nout_result, d_final_oatt4_llinenum, d_nout_result);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_reduce2 = std::clock();


    hipMemcpy( &final_nout_result, d_final_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt4_llinenum.data(), d_final_oatt4_llinenum, 6001215 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }











    hipFree( d_iatt4_llinenum);
    hipFree( d_nout_result);
    hipFree( d_oatt4_llinenum);
    hipFree( d_aht2);  ///
    hipFree( d_final_nout_result);  ///
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", final_nout_result);  ///
    if((final_nout_result > 6001215)) {  ///
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < final_nout_result)); pv += 1) {  ///
        printf("l_linenumber: ");
        printf("%8i", oatt4_llinenum[pv]);
        printf("  ");
        printf("\n");
    }
    if((final_nout_result > 10)) {  ///
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();


    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_reduce1", (stop_krnl_reduce1 - start_krnl_reduce1) / (double) (CLOCKS_PER_SEC / 1000) );  ///
    printf ( "%32s: %6.1f ms\n", "krnl_reduce2", (stop_krnl_reduce2 - start_krnl_reduce2) / (double) (CLOCKS_PER_SEC / 1000) );  ///
    printf("</timing>\n");
}
