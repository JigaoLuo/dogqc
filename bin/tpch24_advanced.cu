#include "hip/hip_runtime.h"
/// My Query 24
/// select l_linenumber --> l_linenumber is the 4th attribute in lineitem table
/// from lineitem
/// group by l_linenumber
#include <unordered_set>  ///
#include <cassert>  ///
#include <cstring>  ///

#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct apayl2 {
    int att4_llinenum;
};

constexpr int SHARED_MEMORY_SIZE = 49152;  /// Total amount of shared memory per block:       49152 bytes

__global__ void krnl_lineitem1(
    int* iatt4_llinenum, int* nout_result, int* oatt4_llinenum) {

    /// local block memory cache : ONLY FOR A BLOCK'S THREADS!!!
    const int HT_SIZE = 128;
    __shared__ agg_ht<apayl2> aht2[HT_SIZE];  ///
    const int shared_memory_usage = sizeof(aht2);
    assert(shared_memory_usage <= SHARED_MEMORY_SIZE);  /// Check stuff fits into shared memory in a SM.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        /// Allow only one print here.
        printf("Shared memory usage: %d / %d bytes.\n", shared_memory_usage, SHARED_MEMORY_SIZE);
    }

    {
        /// Init hash table in shared memory.
        int ht_index;
        unsigned loopVar = threadIdx.x;  ///
        unsigned step = blockDim.x;  ///
        while(loopVar < HT_SIZE) {
            ht_index = loopVar;
            aht2[ht_index].lock.init();
            aht2[ht_index].hash = HASH_EMPTY;
            loopVar += step;
        }
    }

    __syncthreads();

    {
        /// The first old kenrel
        int att4_llinenum;

        int tid_lineitem1 = 0;
        unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned step = (blockDim.x * gridDim.x);
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_lineitem1 = loopVar;
            active = (loopVar < 6001215);
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
                att4_llinenum = iatt4_llinenum[tid_lineitem1];
            }
            // -------- aggregation (opId: 2) --------
            int bucket = 0;
            if(active) {
                uint64_t hash2 = 0;
                hash2 = 0;
                if(active) {
                    hash2 = hash ( (hash2 + ((uint64_t)att4_llinenum)));
                }
                apayl2 payl;
                payl.att4_llinenum = att4_llinenum;
                int bucketFound = 0;
                int numLookups = 0;
                while(!(bucketFound)) {
                    bucket = hashAggregateGetBucket ( aht2, HT_SIZE, hash2, numLookups, &(payl));  ///
                    apayl2 probepayl = aht2[bucket].payload;
                    bucketFound = 1;
                    bucketFound &= ((payl.att4_llinenum == probepayl.att4_llinenum));
                }
            }
            if(active) {
            }
            loopVar += step;
        }
    }

    __syncthreads();  ///

    {
        /// The second old kernel
        int att4_llinenum;
        unsigned warplane = (threadIdx.x % 32);
        unsigned prefixlanes = (0xffffffff >> (32 - warplane));
        int tid_aggregation2 = 0;
//        unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned loopVar = threadIdx.x;  ///
//        unsigned step = (blockDim.x * gridDim.x);
        unsigned step = blockDim.x;  ///
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_aggregation2 = loopVar;
            active = (loopVar < HT_SIZE);  ///
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
            }
            // -------- scan aggregation ht (opId: 2) --------
            if(active) {
                active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
            }
            if(active) {
                apayl2 payl = aht2[tid_aggregation2].payload;
                att4_llinenum = payl.att4_llinenum;
            }
            if(active) {
            }
            // -------- projection (no code) (opId: 3) --------
            // -------- materialize (opId: 4) --------
            int wp;
            int writeMask;
            int numProj;
            writeMask = __ballot_sync(ALL_LANES,active);
            numProj = __popc(writeMask);
            if((warplane == 0)) {
                wp = atomicAdd(nout_result, numProj);
            }
            wp = __shfl_sync(ALL_LANES,wp,0);
            wp = (wp + __popc((writeMask & prefixlanes)));
            if(active) {
                oatt4_llinenum[wp] = att4_llinenum;
            }
            loopVar += step;
        }
    }
}

int main() {
    int* iatt4_llinenum;
    size_t filesize = get_file_size( "mmdb/lineitem_l_linenumber" );  ///
    hipHostMalloc((void**)&iatt4_llinenum, filesize - 8 /* 8: the meta: size of file in 8bytes*/);  /// host pinned
    read_file("mmdb/lineitem_l_linenumber", (void*)iatt4_llinenum );  ///

    int nout_result;
    /// std::vector < int > oatt4_llinenum(6001215);
    int* oatt4_llinenum;  ///
    hipHostMalloc((void**)&oatt4_llinenum, 6001215 * sizeof(int));  /// host pinned

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt4_llinenum;
    hipMalloc((void**) &d_iatt4_llinenum, 6001215* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt4_llinenum;
    hipMalloc((void**) &d_oatt4_llinenum, 6001215* sizeof(int) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt4_llinenum, iatt4_llinenum, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem1<<<gridsize, blocksize>>>(d_iatt4_llinenum, d_nout_result, d_oatt4_llinenum);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }
    std::clock_t stop_totalKernelTime0 = std::clock();

    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    /// hipMemcpy( oatt4_llinenum.data(), d_oatt4_llinenum, 6001215 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt4_llinenum, d_oatt4_llinenum, 6001215 * sizeof(int), hipMemcpyDeviceToHost);  ///
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt4_llinenum);
    hipFree( d_nout_result);
    hipFree( d_oatt4_llinenum);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 6001215)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("l_linenumber: ");
        printf("%8i", oatt4_llinenum[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();

    /// My Reduce
    std::cout << "MY REDUCE ON CPU (single-cpu-threaded)" << std::endl;
    std::clock_t start_cpu_reduce = std::clock();
    std::unordered_set<int> ht;
    for ( int pv = 0; (pv < nout_result); pv += 1 ) {
        ht.emplace(oatt4_llinenum[pv]);
    }
    for (const auto& ele : ht) {
        printf("l_linenumber: ");
        printf("%8i", ele);
        printf("  ");
        printf("\n");
    }
    std::clock_t stop_cpu_reduce = std::clock();

    hipHostFree( iatt4_llinenum );  ///
    hipHostFree( oatt4_llinenum );  ///
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free host! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free host")
        }
    }


    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "reduce on CPU (single-cpu-threaded)", (stop_cpu_reduce - start_cpu_reduce) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
