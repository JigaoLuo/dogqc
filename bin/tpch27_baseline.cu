#include "hip/hip_runtime.h"
/// select l_orderkey, count(*)
/// from lineitem
/// group by l_orderkey
/// order by l_orderkey
#include <map>

#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct apayl2 {
    int att2_lorderke;
};

__global__ void krnl_lineitem1(
    int* iatt2_lorderke, agg_ht<apayl2>* aht2, int* agg1) {
    int att2_lorderke;

    int tid_lineitem1 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_lineitem1 = loopVar;
        active = (loopVar < 6001215);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
            att2_lorderke = iatt2_lorderke[tid_lineitem1];
        }
        // -------- aggregation (opId: 2) --------
        int bucket = 0;
        if(active) {
            uint64_t hash2 = 0;
            hash2 = 0;
            if(active) {
                hash2 = hash ( (hash2 + ((uint64_t)att2_lorderke)));
            }
            apayl2 payl;
            payl.att2_lorderke = att2_lorderke;
            int bucketFound = 0;
            int numLookups = 0;
            while(!(bucketFound)) {
                bucket = hashAggregateGetBucket ( aht2, 12002430, hash2, numLookups, &(payl));
                apayl2 probepayl = aht2[bucket].payload;
                bucketFound = 1;
                bucketFound &= ((payl.att2_lorderke == probepayl.att2_lorderke));
            }
        }
        if(active) {
            atomicAdd(&(agg1[bucket]), ((int)1));
        }
        loopVar += step;
    }

}

__global__ void krnl_aggregation2(
    agg_ht<apayl2>* aht2, int* agg1, int* nout_result, int* oatt2_lorderke, int* oatt1_countlor) {
    int att2_lorderke;
    int att1_countlor;
    unsigned warplane = (threadIdx.x % 32);
    unsigned prefixlanes = (0xffffffff >> (32 - warplane));

    int tid_aggregation2 = 0;
    unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
    unsigned step = (blockDim.x * gridDim.x);
    unsigned flushPipeline = 0;
    int active = 0;
    while(!(flushPipeline)) {
        tid_aggregation2 = loopVar;
        active = (loopVar < 12002430);
        // flush pipeline if no new elements
        flushPipeline = !(__ballot_sync(ALL_LANES,active));
        if(active) {
        }
        // -------- scan aggregation ht (opId: 2) --------
        if(active) {
            active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
        }
        if(active) {
            apayl2 payl = aht2[tid_aggregation2].payload;
            att2_lorderke = payl.att2_lorderke;
        }
        if(active) {
            att1_countlor = agg1[tid_aggregation2];
        }
        // -------- projection (no code) (opId: 3) --------
        // -------- materialize (opId: 4) --------
        int wp;
        int writeMask;
        int numProj;
        writeMask = __ballot_sync(ALL_LANES,active);
        numProj = __popc(writeMask);
        if((warplane == 0)) {
            wp = atomicAdd(nout_result, numProj);
        }
        wp = __shfl_sync(ALL_LANES,wp,0);
        wp = (wp + __popc((writeMask & prefixlanes)));
        if(active) {
            oatt2_lorderke[wp] = att2_lorderke;
            oatt1_countlor[wp] = att1_countlor;
        }
        loopVar += step;
    }

}

int main() {
    int* iatt2_lorderke;
    iatt2_lorderke = ( int*) map_memory_file ( "mmdb/lineitem_l_orderkey" );

    int nout_result;
    std::vector < int > oatt2_lorderke(6001215);
    std::vector < int > oatt1_countlor(6001215);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt2_lorderke;
    hipMalloc((void**) &d_iatt2_lorderke, 6001215* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt2_lorderke;
    hipMalloc((void**) &d_oatt2_lorderke, 6001215* sizeof(int) );
    int* d_oatt1_countlor;
    hipMalloc((void**) &d_oatt1_countlor, 6001215* sizeof(int) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    agg_ht<apayl2>* d_aht2;
    hipMalloc((void**) &d_aht2, 12002430* sizeof(agg_ht<apayl2>) );
    {
        int gridsize=920;
        int blocksize=128;
        initAggHT<<<gridsize, blocksize>>>(d_aht2, 12002430);
    }
    int* d_agg1;
    hipMalloc((void**) &d_agg1, 12002430* sizeof(int) );
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_agg1, 0, 12002430);
    }
    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt2_lorderke, iatt2_lorderke, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    std::clock_t start_krnl_lineitem11 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_lineitem1<<<gridsize, blocksize>>>(d_iatt2_lorderke, d_aht2, d_agg1);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_lineitem11 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }

    std::clock_t start_krnl_aggregation22 = std::clock();
    {
        int gridsize=920;
        int blocksize=128;
        krnl_aggregation2<<<gridsize, blocksize>>>(d_aht2, d_agg1, d_nout_result, d_oatt2_lorderke, d_oatt1_countlor);
    }
    hipDeviceSynchronize();
    std::clock_t stop_krnl_aggregation22 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_aggregation2! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_aggregation2")
        }
    }

    std::clock_t stop_totalKernelTime0 = std::clock();
    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt2_lorderke.data(), d_oatt2_lorderke, 6001215 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_countlor.data(), d_oatt1_countlor, 6001215 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt2_lorderke);
    hipFree( d_aht2);
    hipFree( d_agg1);
    hipFree( d_nout_result);
    hipFree( d_oatt2_lorderke);
    hipFree( d_oatt1_countlor);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 6001215)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("l_orderkey: ");
        printf("%8i", oatt2_lorderke[pv]);
        printf("  ");
        printf("count_l_orderkey: ");
        printf("%8i", oatt1_countlor[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...]\n");
    }
    printf("\n");

    /// 40 sorted output
    std::map<int, int> ht;
    for ( int pv = 0; pv < nout_result; pv += 1) {
        ht.emplace(oatt2_lorderke[pv], oatt1_countlor[pv]);
    }
    for ( int pv = 0, auto it = ht.begin(); ((pv < 40) && (pv < ht.size())); pv += 1) {
        printf("l_orderkey: ");
        printf("%8i", it.first);
        printf("  ");
        printf("count_l_orderkey: ");
        printf("%8i", it.second);
        printf("  ");
        printf("\n");
        std::advance(it, 1);
    }
    if((ht.size() > 40)) {
        printf("[...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "krnl_lineitem1", (stop_krnl_lineitem11 - start_krnl_lineitem11) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "krnl_aggregation2", (stop_krnl_aggregation22 - start_krnl_aggregation22) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
