#include "hip/hip_runtime.h"
/// My Query 27
/// select l_orderkey, count(*) --> l_orderkey is the 1st attribute in lineitem table
/// from lineitem
/// group by l_orderkey
/// order by l_orderkey
#include <map>
#include <cassert>

#include <list>
#include <unordered_map>
#include <vector>
#include <iostream>
#include <ctime>
#include <limits.h>
#include <float.h>
#include "../dogqc/include/csv.h"
#include "../dogqc/include/util.h"
#include "../dogqc/include/mappedmalloc.h"
#include "../dogqc/include/util.cuh"
#include "../dogqc/include/hashing.cuh"
struct apayl2 {
    int att2_lorderke;
};

constexpr int SHARED_MEMORY_SIZE = 49152;  /// Total amount of shared memory per block:       49152 bytes


__global__ void krnl_lineitem1(
    int* iatt2_lorderke, int* nout_result, int* oatt2_lorderke, int* oatt1_countlor) {  ///

    /// local block memory cache : ONLY FOR A BLOCK'S THREADS!!!
    const int HT_SIZE = 1280;
    __shared__ agg_ht<apayl2> aht2[HT_SIZE];  ///
    __shared__ int agg1[HT_SIZE];  ///
    const int shared_memory_usage = sizeof(aht2) + sizeof(agg1);
    assert(shared_memory_usage <= SHARED_MEMORY_SIZE);  /// Check stuff fits into shared memory in a SM.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        /// Allow only one print here.
        printf("Shared memory usage: %d / %d bytes.\n", shared_memory_usage, SHARED_MEMORY_SIZE);
    }

    {
        /// Init hash table in shared memory.
        int ht_index;
        unsigned loopVar = threadIdx.x;  ///
        unsigned step = blockDim.x;  ///
        while(loopVar < HT_SIZE) {
            ht_index = loopVar;
            aht2[ht_index].lock.init();
            aht2[ht_index].hash = HASH_EMPTY;
            loopVar += step;
        }
    }

    {
        /// Init array in shared memory.
        int index;
        unsigned loopVar = threadIdx.x;  ///
        unsigned step = blockDim.x;  ///
        while(loopVar < HT_SIZE) {
            index = loopVar;
            agg1[index] = 0;
            loopVar += step;
        }
    }

    __syncthreads();

    {
        /// The first old kenrel
        int att2_lorderke;

        int tid_lineitem1 = 0;
        unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned step = (blockDim.x * gridDim.x);
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_lineitem1 = loopVar;
            active = (loopVar < 6001215);
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
                att2_lorderke = iatt2_lorderke[tid_lineitem1];
            }
            // -------- aggregation (opId: 2) --------
            int bucket = 0;
            if(active) {
                uint64_t hash2 = 0;
                hash2 = 0;
                if(active) {
                    hash2 = hash ( (hash2 + ((uint64_t)att2_lorderke)));
                }
                apayl2 payl;
                payl.att2_lorderke = att2_lorderke;
                int bucketFound = 0;
                int numLookups = 0;
                while(!(bucketFound)) {
                    bucket = hashAggregateGetBucket ( aht2, HT_SIZE, hash2, numLookups, &(payl));  ///
                    apayl2 probepayl = aht2[bucket].payload;
                    bucketFound = 1;
                    bucketFound &= ((payl.att2_lorderke == probepayl.att2_lorderke));
                }
            }
            if(active) {
                atomicAdd(&(agg1[bucket]), ((int)1));
            }
            loopVar += step;
        }
    }

    __syncthreads();  ///

    {
        /// The second old kernel
        int att2_lorderke;
        int att1_countlor;
        unsigned warplane = (threadIdx.x % 32);
        unsigned prefixlanes = (0xffffffff >> (32 - warplane));

        int tid_aggregation2 = 0;
        /// unsigned loopVar = ((blockIdx.x * blockDim.x) + threadIdx.x);
        unsigned loopVar = threadIdx.x;  ///
        /// unsigned step = (blockDim.x * gridDim.x);
        unsigned step = blockDim.x;  ///
        unsigned flushPipeline = 0;
        int active = 0;
        while(!(flushPipeline)) {
            tid_aggregation2 = loopVar;
            active = (loopVar < HT_SIZE);  ///
            // flush pipeline if no new elements
            flushPipeline = !(__ballot_sync(ALL_LANES,active));
            if(active) {
            }
            // -------- scan aggregation ht (opId: 2) --------
            if(active) {
                active &= ((aht2[tid_aggregation2].lock.lock == OnceLock::LOCK_DONE));
            }
            if(active) {
                apayl2 payl = aht2[tid_aggregation2].payload;
                att2_lorderke = payl.att2_lorderke;
            }
            if(active) {
                att1_countlor = agg1[tid_aggregation2];
            }
            // -------- projection (no code) (opId: 3) --------
            // -------- materialize (opId: 4) --------
            int wp;
            int writeMask;
            int numProj;
            writeMask = __ballot_sync(ALL_LANES,active);
            numProj = __popc(writeMask);
            if((warplane == 0)) {
                wp = atomicAdd(nout_result, numProj);
            }
            wp = __shfl_sync(ALL_LANES,wp,0);
            wp = (wp + __popc((writeMask & prefixlanes)));
            if(active) {
                oatt2_lorderke[wp] = att2_lorderke;
                oatt1_countlor[wp] = att1_countlor;
            }
            loopVar += step;
        }
    }

}

int main() {
    int* iatt2_lorderke;
    iatt2_lorderke = ( int*) map_memory_file ( "mmdb/lineitem_l_orderkey" );

    int nout_result;
    std::vector < int > oatt2_lorderke(6001215);
    std::vector < int > oatt1_countlor(6001215);

    // wake up gpu
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in wake up gpu! " << hipGetErrorString( err ) << std::endl;
            ERROR("wake up gpu")
        }
    }

    int* d_iatt2_lorderke;
    hipMalloc((void**) &d_iatt2_lorderke, 6001215* sizeof(int) );
    int* d_nout_result;
    hipMalloc((void**) &d_nout_result, 1* sizeof(int) );
    int* d_oatt2_lorderke;
    hipMalloc((void**) &d_oatt2_lorderke, 6001215* sizeof(int) );
    int* d_oatt1_countlor;
    hipMalloc((void**) &d_oatt1_countlor, 6001215* sizeof(int) );
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda malloc! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda malloc")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    {
        int gridsize=920;
        int blocksize=128;
        initArray<<<gridsize, blocksize>>>(d_nout_result, 0, 1);
    }
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda mallocHT! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda mallocHT")
        }
    }


    // show memory usage of GPU
    {   size_t free_byte ;
        size_t total_byte ;
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
        if ( hipSuccess != cuda_status ) {
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        fprintf(stderr, "Memory %.1f / %.1f GB\n",
                used_db/(1024*1024*1024), total_db/(1024*1024*1024) );
        fflush(stdout);
    }

    hipMemcpy( d_iatt2_lorderke, iatt2_lorderke, 6001215 * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy in! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy in")
        }
    }

    std::clock_t start_totalKernelTime0 = std::clock();
    {
        int gridsize=920;
        ///int blocksize=128;
        int blocksize=1280;
        krnl_lineitem1<<<(6001215 + 127) / blocksize, blocksize>>>(d_iatt2_lorderke, d_nout_result, d_oatt2_lorderke, d_oatt1_countlor);
    }
    hipDeviceSynchronize();
    std::clock_t stop_totalKernelTime0 = std::clock();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in krnl_lineitem1! " << hipGetErrorString( err ) << std::endl;
            ERROR("krnl_lineitem1")
        }
    }

    hipMemcpy( &nout_result, d_nout_result, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt2_lorderke.data(), d_oatt2_lorderke, 6001215 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy( oatt1_countlor.data(), d_oatt1_countlor, 6001215 * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda memcpy out! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda memcpy out")
        }
    }

    hipFree( d_iatt2_lorderke);
    hipFree( d_nout_result);
    hipFree( d_oatt2_lorderke);
    hipFree( d_oatt1_countlor);
    hipDeviceSynchronize();
    {
        hipError_t err = hipGetLastError();
        if(err != hipSuccess) {
            std::cerr << "Cuda Error in cuda free! " << hipGetErrorString( err ) << std::endl;
            ERROR("cuda free")
        }
    }

    std::clock_t start_finish3 = std::clock();
    printf("\nResult: %i tuples\n", nout_result);
    if((nout_result > 6001215)) {
        ERROR("Index out of range. Output size larger than allocated with expected result number.")
    }
    for ( int pv = 0; ((pv < 10) && (pv < nout_result)); pv += 1) {
        printf("l_orderkey: ");
        printf("%8i", oatt2_lorderke[pv]);
        printf("  ");
        printf("count_l_orderkey: ");
        printf("%8i", oatt1_countlor[pv]);
        printf("  ");
        printf("\n");
    }
    if((nout_result > 10)) {
        printf("[...unsorted...]\n");
    }
    printf("\n");
    std::clock_t stop_finish3 = std::clock();

    /// 40 sorted output
    std::map<int, int> ht;
    for ( int pv = 0; pv < nout_result; pv += 1) {
        ht.emplace(oatt2_lorderke[pv], oatt1_countlor[pv]);
    }
    auto it = ht.begin();
    for ( int pv = 0; ((pv < 25) && (pv < ht.size())); pv += 1) {
        printf("l_orderkey: ");
        printf("%8i", it->first);
        printf("  ");
        printf("count_l_orderkey: ");
        printf("%8i", it->second);
        printf("  ");
        printf("\n");
        std::advance(it, 1);
    }
    if((ht.size() > 25)) {
        printf("[...sorted...]\n");
    }
    printf("\n");

    printf("<timing>\n");
    printf ( "%32s: %6.1f ms\n", "finish", (stop_finish3 - start_finish3) / (double) (CLOCKS_PER_SEC / 1000) );
    printf ( "%32s: %6.1f ms\n", "totalKernelTime", (stop_totalKernelTime0 - start_totalKernelTime0) / (double) (CLOCKS_PER_SEC / 1000) );
    printf("</timing>\n");
}
